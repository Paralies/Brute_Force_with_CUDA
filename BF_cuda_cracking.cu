#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
			       hipGetErrorString(e), \
			       __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif

int main() {

    std::ofstream logFile;
    logFile.open("bruteForce.log");

    std::ofstream passwordFile;
    passwordFile.open("password.txt");

    CharSet passwordBF;
    passwordBF.make_password();

    for(auto i : passwordBF.passwordsHash) {
        passwordFile << i << std::endl;
        //std::cout << i << std::endl;
    }
    
    logFile.close();
    passwordFile.close();
}